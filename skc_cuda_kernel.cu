#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>


#define input_centric_backward      // input-centric backward computing.
#define enforce_atomic              // for input-centric backward correctness guarantee

template <typename scalar_t>
__global__ void skc_cuda_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> new_tensor,
    int batch_size,
    int input_channel,
    int input_height,
    int input_width,
    int output_channel,
    int input_unit_dim,
    int stride
);

template <typename scalar_t>
__global__ void skc_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_weights,
    int batch_size,
    int input_channel,
    int height,
    int width,
    int output_channel,
    int input_unit_dim,
    int stride
);
////////////////////////////////////////////
// foward pass
////////////////////////////////////////////
std::vector<torch::Tensor> skc_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor new_tensor,
    int stirde
) {

    // input: batch_size * input_channel * input_width * input_height.
    const int batch_size = input.size(0);
    const int input_channel = input.size(1);
    const int input_height = input.size(2);
    const int input_width = input.size(3);

    // output: batch_size * opt_channel * input_width * input_height.
    const int output_channel = weights.size(0);

    // weight: output_channel * input_units_dim * 1.
    const int input_unit_dim = weights.size(1); 

    // new tensor for output.
    const int threads = 1024;   //һ��block��1024���߳�
    const int blocks = (batch_size * output_channel * input_width * input_height + threads - 1) / threads;   //����һ��grid��Ҫ���ٸ�block,������ȡ��
	
	/*AT_DISPATCH_FLOATING_TYPES����꣬ʵ���˶�̬�ַ����ƣ�dynamic dispatch����������������ʱ����������������ֵ���ͣ�ȥ����֮ǰCUDA kernelģ�麯����Ҫʵ����Ϊ���ֺ�����
	�������Ҫ�����������ͣ������������ַ�����Ϣ��һ��lambda��������������������src.scalar_type()��ȡ��lambda�������Ǻ������CUDA kernel�������֡�
	�������ڴ�ATen Tensor�л�ȡĳһ��������ָ����õ���<<< >>>��һд������kernel��*/
    AT_DISPATCH_FLOATING_TYPES(input.type(), "skc_forward_cuda", ([&] {
                                skc_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                                    input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                    weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    new_tensor.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                    batch_size,
                                    input_channel,
                                    input_height,
                                    input_width,
                                    output_channel,
                                    input_unit_dim,
                                    stirde
                                );
                            }));
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {new_tensor};
}
//The argument torch::RestrictPtrTraits indicates that the __restrict__ keyword must be used.
template <typename scalar_t>
__global__ void skc_cuda_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> new_tensor,
    int batch_size,
    int input_channel,
    int input_height,
    int input_width,
    int output_channel,
    int input_unit_dim,
    int stride
) {
  const int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int g_dim = batch_size * output_channel * input_width * input_height;
  const int item_size_dim = output_channel * input_height * input_width;
  const int feature_map_dim = input_height * input_width;

  const int item_idx = g_idx / item_size_dim;
  const int item_channel_idx =  (g_idx - item_idx * item_size_dim) / feature_map_dim;
  const int item_feat_y_idx = (g_idx - item_idx * item_size_dim - item_channel_idx * feature_map_dim) / input_width;
  const int item_feat_x_idx = (g_idx - item_idx * item_size_dim - item_channel_idx * feature_map_dim) % input_width;
  const int b = item_idx;
  const int c = item_channel_idx;
  const int y = item_feat_y_idx;
  const int x = item_feat_x_idx;
  
  const int input_c_start = __float2int_rd(item_channel_idx * stride) % input_channel;
  const int input_c_end = (input_c_start + input_unit_dim) % input_channel;
  const int input_x = x;
  const int input_y = y;

//   new_tensor[b][c][y][x] = 5;
//   printf("gid: %d, total thread: %d\n", g_idx, g_dim);
  if (g_idx < g_dim) {
        float tmp = 0;
        // printf("input_c_start, %d, input_c_end, %d\n", input_c_start, input_c_end);
        if (input_c_start < input_c_end)
            for(int c_input_d = input_c_start; c_input_d < input_c_end; c_input_d++){
                tmp += input[b][c_input_d][input_y][input_x] * weights[c][c_input_d - input_c_start];
            }
        else
        {
            for(int c_input_d = input_c_start; c_input_d < input_channel; c_input_d++){
                tmp += input[b][c_input_d][input_y][input_x] * weights[c][c_input_d - input_c_start];
            }
            for(int c_input_d = 0; c_input_d < input_c_end; c_input_d++){
                tmp += input[b][c_input_d][input_y][input_x] * weights[c][c_input_d + input_channel - input_c_start];
            } 
        }
        new_tensor[b][c][y][x] = tmp;
        // printf("gid: %d, new tensor (%d, %d, %d, %d) --- %f\n", g_idx, b, c, y, x, new_tensor[0][0][0][0]);
  }
}

#ifdef input_centric_backward
std::vector<torch::Tensor> skc_cuda_backward(
        torch::Tensor d_output,
        torch::Tensor input,
        torch::Tensor weights,
        torch::Tensor d_input,
        torch::Tensor d_weights,
        int stride
    ) {
    
        // input: batch_size * input_channel * input_width * input_height.
        const int batch_size = d_output.size(0);
        const int output_channel = d_output.size(1);
        const int height = d_output.size(2);
        const int width = d_output.size(3);
    
        // output: batch_size * opt_channel * input_width * input_height.
        const int input_channel = d_input.size(1);
    
        // weight: output_channel * input_units_dim * 1.
        const int input_unit_dim = weights.size(1); 
    
        const int threads = 1024;
        const int blocks = (batch_size * input_channel * width * height + threads - 1) / threads; 
    
        AT_DISPATCH_FLOATING_TYPES(d_output.type(), "skc_backward_cuda", ([&] {
                                skc_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
                                        d_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                        weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                        d_input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                        d_weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                        batch_size,
                                        input_channel,
                                        height,
                                        width,
                                        output_channel,
                                        input_unit_dim,
                                        stride
                                    );
                                }));
        // check for error
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        return {d_input, d_weights};
    }
    
template <typename scalar_t>
__global__ void skc_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_weights,
    int batch_size,
    int input_channel,
    int height,
    int width,
    int output_channel,
    int input_unit_dim,
    int stride
) {
    const int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int g_dim = batch_size * input_channel * width * height;

//   printf("gid: %d, total thread: %d\n", g_idx, g_dim);
    if (g_idx < g_dim) {
        const int item_size_dim = input_channel * height * width;
        const int feature_map_dim = height * width;
        const int item_idx = g_idx / item_size_dim;
        const int item_channel_idx =  (g_idx - item_idx * item_size_dim) / feature_map_dim;
        const int b = item_idx;
        const int y = (g_idx - item_idx * item_size_dim - item_channel_idx * feature_map_dim) / width;
        const int x = (g_idx - item_idx * item_size_dim - item_channel_idx * feature_map_dim) % width;
        int cid;
        int const_term = stride;

        for (int v_cid = item_channel_idx; true; v_cid += input_channel){
            
            #ifdef debug
            if (item_idx == 0 && x == 0 && y == x && item_channel_idx == 3)
            {
                printf("v_cid: %d\n", v_cid);
            }
            #endif

            int output_start_idx = v_cid / const_term;
            int output_start_offset = v_cid % const_term;

            int output_end_idx = 0;
            int output_end_offset = 0;
            
            if (v_cid < input_unit_dim){
                output_end_idx = 0;
                output_end_offset = v_cid;
            } 
            else{
                output_end_idx = __float2int_rd((v_cid - input_unit_dim) * 1.0f / const_term) + 1; //���ƾ�������((n-f)/s+1),��ʾ�������ͨ������ڼ���������
                output_end_offset = v_cid - const_term * output_end_idx; 
            }  //��������˳��ϲ���s����ʾ�����˵ĵ�һ��ͨ����λ�ã� v_cid��ʾĿǰ����ͨ��λ�ã� v_cid-�þ����˵ĵ�һ��ͨ����λ�õ���v_cid�ڸþ����˵�һ��ͨ������Ծ���

            #ifdef debug
            if (item_idx == 0 && x == 0 && y == x && item_channel_idx == 3)
            printf("opt_start_idx: %d\nopt_start_offset: %d\nopt_end_idx: %d\nopt_end_offset: %d\n\n", output_start_idx, output_start_offset, output_end_idx, output_end_offset);
            #endif

            if (output_start_idx >= output_channel && output_end_idx >= output_channel) break;
						
            cid = v_cid % input_channel;
            if (output_start_idx == output_end_idx){
                d_input[b][cid][y][x] += weights[output_start_idx][output_start_offset] * d_output[b][output_start_idx][y][x];

                #ifdef enforce_atomic
                atomicAdd((float*)&d_weights[output_start_idx][output_start_offset], input[b][cid][y][x] * d_output[b][output_start_idx][y][x]);
                #else
                d_weights[output_start_idx][output_start_offset] += input[b][cid][y][x] * d_output[b][output_start_idx][y][x];
                #endif
            }
            else{
				for(int chout = output_end_idx; chout <= output_start_idx; chout++)
				{	
					if(chout < output_channel)
					{
						int chout_offset = (chout - output_end_idx) * const_term;
						d_input[b][cid][y][x] += weights[chout][output_end_offset - chout_offset] * d_output[b][chout][y][x];
						
						#ifdef enforce_atomic
						atomicAdd((float*)&d_weights[chout][output_end_offset - chout_offset], input[b][cid][y][x] * d_output[b][chout][y][x]);
						#else
						d_weights[chout][output_end_offset - chout_offset] += input[b][cid][y][x] * d_output[b][chout][y][x];
						#endif						
					}
				}
            }

        }
    }
}
#endif


